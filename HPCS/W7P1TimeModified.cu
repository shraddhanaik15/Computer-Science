
#include"hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>
#include<time.h>


__global__ void Toggle(char *a, char *b, int n)
{
	
	int tid;
	tid = threadIdx.x;
	if(a[tid]>='A' && a[tid]<='Z')
	{
	b[tid]=a[tid]+32;
	}
	else
	{
	b[tid]=a[tid]-32;
	}
}

int main()
{
	clock_t t;
	t = clock();
	char a[100],b[100];
	int i,n,size;
	char *d_a, *d_b;
	

	printf("\nEnter the string\n");
	scanf("%s", a);

	n = strlen(a);
	printf("\nNo of charcaters is\t%d", n);

	
	size = sizeof(char);
	printf("\nSize is \t%d\n", size);

	hipMalloc((void **)&d_a,n*size);
	hipMalloc((void **)&d_b,n*size);
	

	hipMemcpy(d_a,a,n*size,hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start,0);

	Toggle<<<1,n>>>(d_a,d_b,n);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms,start,stop);
	printf("\nTime to calculate results inside GPU is: %fms\n",elapsed_time_ms);


	hipMemcpy(b,d_b,n*size,hipMemcpyDeviceToHost);

	
		printf("\nToggled string is \n");
		for(i=0;i<n;i++)
		printf("%c",b[i]);
	
	hipFree(d_a);
	hipFree(d_b);
	t = clock()-t;
	double time_taken;
	time_taken = ((double)t)/CLOCKS_PER_SEC;
	printf("\nEntire program took %f seconds to execute\n", time_taken);

	return 0;


}