#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<string.h>

__global__ void RevWrd(char* a,char* c,int n,int* pos)
{
	int i,k;
	int tid = threadIdx.x;
	int strIn = pos[tid];
	int nstrIn = pos[tid + 1];
	if (nstrIn == 0)
	{
		nstrIn = n;
	}
	k = nstrIn - 2;
	printf("i=%d ,j=%d", strIn, nstrIn);
	for (i = strIn; i < nstrIn;i++)
	{
		c[k--] = a[i];
	}
}
void MyWords(char* arr, int* n,int len)
{
	int i=0,k=0;
	n[k++] = 0;
	while (i != len)
	{
		if (arr[i++] == ' ')
		{
			n[k++] = i;
		}
	}
	n[k] = len;
}
int main(void)
{
	int N = 0, i, sz, strLth;
	char A[100],C[100]; int pos[20];
	char* d_a;char* d_c; int* d_pos;

	printf("Enter no of words in string:\n");
	scanf("%d", &N);
	printf("Enter string:\n");
	scanf(" %[^\n]s", A);
	strLth = strlen(A);
	printf("Strlen %d", strLth);
	MyWords(A, pos,strLth);
	//pos[N + 1] = strLth;
	for (i = 0;i < N+1 ; i++)
	{
		printf(" pos %d =%d\t", i, pos[i]);
	}
	sz = sizeof(char) * strLth;
	hipMalloc((void**)&d_a, sz);
	hipMalloc((void**)&d_pos, N * sizeof(int));
	hipMalloc((void**)&d_c, sz);

	hipMemcpy(d_a, A, sz, hipMemcpyHostToDevice);
	hipMemcpy(d_pos, pos, N * sizeof(int), hipMemcpyHostToDevice);
	
	RevWrd << <1, N >> > (d_a,d_c,strLth,d_pos);

	hipMemcpy(C, d_c, sz, hipMemcpyDeviceToHost);
	printf("%s", C);

	hipFree(d_a);
	hipFree(d_pos);
	hipFree(d_c);
}