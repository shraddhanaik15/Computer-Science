#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void matrixmul(int *a,int *b,int *c,int m,int n,int p)
{
int id = threadIdx.x;
int sum,i,j;
for(i=0;i<m;i++)
{
	sum=0;
	for(j=0;j<p;j++)
	{
	sum+=(a[i*n+j]*b[j*p+id]);
	}
	c[i*p+id]=sum;

}
}

int main(void)
{
	clock_t t;
	t = clock();
	int m,n,a[100], b[100], c[100],p,i,j;
	int size,*d_a,*d_b,*d_c;

	size = sizeof(int);
	printf("Enter value of m, n, p\n");
	scanf("%d%d%d",&m,&n,&p);


	printf("Enter Matrix A\n");
	for(i=0;i<m*n;i++)
	{
	scanf("%d",&a[i]);
	}


	printf("Enter Matrix B\n");
	for(i=0;i<n*p;i++)
	{
	scanf("%d",&b[i]);
	}

	hipMalloc((void **)&d_a,m*n*size);
	hipMalloc((void **)&d_b,p*n*size);
	hipMalloc((void **)&d_c,m*p*size);

	hipMemcpy(d_a,a,m*n*size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,n*p*size,hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start,0);
	
	matrixmul<<<1,p>>>(d_a,d_b,d_c,m,n,p);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms,start,stop);
	printf("\nTime to calculate results inside GPU is: %fms\n",elapsed_time_ms);


	hipMemcpy(c,d_c,size*m*p,hipMemcpyDeviceToHost);

	printf("\nMultiplication is :\n");
	for(i=0;i<m;i++)
	{
	printf("\n");
	for(j=0;j<p;j++)
	{
	printf("%d\t",c[i*m+j]);
	}
	}
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	t = clock()-t;
	double time_taken;
	time_taken = ((double)t)/CLOCKS_PER_SEC;
	printf("\nEntire program took %f seconds to execute\n", time_taken);


	return 0;

}
