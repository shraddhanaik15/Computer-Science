#include "hip/hip_runtime.h"

#include <stdio.h>
#define N 5

__global__ void add(int* a, int* b, int* c)
{
	int tid;
	tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
}


int main(void)
{
	int a[N],b[N],c[N];
	
	int size=sizeof(int);

	int i;
	for(i=0;i<N;i++)
	{
	a[i] = i;
	b[i] = 2*i;
	}
	int *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a,N*size);
	hipMalloc((void **)&d_b,N*size);
	hipMalloc((void **)&d_c,N*size);

	hipMemcpy(d_a,a,N*size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,N*size,hipMemcpyHostToDevice);
	

	add<<<1,N>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c,size*N,hipMemcpyDeviceToHost);

	for(i=0;i<N;i++)
	printf("Sum is %d\n",c[i]);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;

}