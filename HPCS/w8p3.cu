#include"hip/hip_runtime.h"

#include<stdio.h>

__global__ void convert(int *b, int *c, int n)
{
	int tx,ty,i;
	
	tx = threadIdx.x;
	ty = threadIdx.y;

	
	if(tx==ty)
	{
	c[tx*n+ty]=0;
	}
	
	else if(tx>ty)
	{
	int fact = 1;
	for(i=1;i<b[ty*n+tx];i++)
	{
	fact += fact * i;
	}
	c[ty*n+tx]=fact;
	//printf("\nx = %d, y= %d ,fact =%d",tx,ty,fact);
	}

	else if(ty>tx)
	{
	int num=b[ty*n+tx];
	int r=0; 
	while(num>0)
	{
	r+=num%10;
	num=num/10;
	}
	c[ty*n+tx]=r;
	//printf("\nx = %d, y= %d ,num =%d",tx,ty,r);
	}


}


int main(void)
{
	int n,i,j,a[3][3],b[9],c[9];
	int  *d_b,*d_c, size;

	printf("Enter value of n\n");
	scanf("%d", &n);

	printf("Enter the elements of the matrix\n");
	for(i=0;i<n;i++)
	{
	for(j=0;j<n;j++)
	{
	scanf("%d", &a[i][j]);
	}
	}

	/*for(i=0;i<n*n;i++)
	{
	b[i]=a[i][j];
	}*/

	size = sizeof(int);

	hipMalloc((void **)&d_b,n*n*size);
	hipMalloc((void **)&d_c,n*n*size);
	
	hipMemcpy(d_b,a,n*n*size,hipMemcpyHostToDevice);

	dim3 dimgrid(1,1,1);
	dim3 dimblock(n,n,1);
	
	convert<<<dimgrid,dimblock>>>(d_b,d_c,n);

	hipMemcpy(c,d_c,n*n*size,hipMemcpyDeviceToHost);
	printf("\n");
	for(i=0;i<n;i++)
	{
	for(j=0;j<n;j++)
	{
		printf("%d\t", c[i*n+j]);
	}
	printf("\n");
	}
		
	
	hipFree(d_b);
	hipFree(d_c);
	return 0;

}