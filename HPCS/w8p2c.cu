#include"hip/hip_runtime.h"

#include<stdio.h>

__global__ void Add(int *a, int *b, int *c, int n)
{
	int id,i;
	id = threadIdx.x;
		c[id] = a[id]+b[id];
}

int main()
{
	int a[100],b[100],c[100],n,i;
	int size, *d_a, *d_b, *d_c;

	printf("\nEnter N\n");
	scanf("%d", &n);

	printf("\nEnter MATRIX A\n");
	for(i=0;i<n*n;i++)
	{
		scanf("%d", &a[i]);
	}
	

	printf("\nEnter MATRIX B\n");
	for(i=0;i<n*n;i++)
	{
		scanf("%d", &b[i]);
	}

	size = sizeof(int);

	hipMalloc((void **)&d_a,n*n*size);
	hipMalloc((void **)&d_b,n*n*size);
	hipMalloc((void **)&d_c,n*n*size);

	hipMemcpy(d_a,a,n*n*size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,n*n*size,hipMemcpyHostToDevice);

	Add<<<1,n*n>>>(d_a,d_b,d_c,n);

	hipMemcpy(c,d_c,size*n*n,hipMemcpyDeviceToHost);

	printf("Addition of rows\n");
	for(i=0;i<n*n;i++)
	{
		printf("%d\t", c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;


}