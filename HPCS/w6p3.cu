#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void swap(int *a, int n)
{

	int temp;

	int i = blockIdx.x  * blockDim.x + 0;
	int j = blockIdx.x  * blockDim.x + 1;

	temp = a[i];
	a[i]= a[j];
	a[j]=temp;

}

int main(void)
{
	int n,a[100],i;
	printf("Enter no of elements in the array\n");
	scanf("%d", &n);
	printf("Enter the array elements\n");
	for(i=0;i<n;i++)
	{
	scanf("%d", &a[i]);
	}

	int *d_a,size;

	size = sizeof(int);

	hipMalloc((void **)&d_a,n*size);
	
	
	hipMemcpy(d_a,a,n*size,hipMemcpyHostToDevice);

	dim3 dimgrid(n/2,1,1);
	dim3 dimblock(2,1,1);

	swap<<<dimgrid,dimblock>>>(d_a,n);

	hipMemcpy(a,d_a,n*size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
	{
	printf("%d\n", a[i]);
	}

	hipFree(d_a);
	

	return 0;
}