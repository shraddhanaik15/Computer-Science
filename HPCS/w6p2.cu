
#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void convert(int *a, int *b,int n)
{
int tid,octalnum=0,i=1,num;
tid = threadIdx.x;

num=a[tid];

while(num>0)
{
 octalnum+=(num%8)*i;
 num=num/8;
 i=i*10;
 }
 b[tid]=octalnum;
}

int main(void)
{
	
	int a[100],n,i,b[100],size;
	printf("Enter the no of decimal values to be converted to Octal\n");
	scanf("%d", &n);
	printf("Enter the Decimal values\n");
	for(i=0;i<n;i++)
	{
	scanf("%d", &a[i]);
	}

	int *d_a, *d_b;
	size = sizeof(int);

	hipMalloc((void **)&d_a,n*size);
	hipMalloc((void **)&d_b,n*size);

	hipMemcpy(d_a,a,n*size,hipMemcpyHostToDevice);

	convert<<<1,n>>>(d_a,d_b,n);

	hipMemcpy(b,d_b,n*size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
	{
	printf("%d\n", b[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	return 0;
	
}