#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#define MAX 50
#define Block_Size 2

__global__ void RevWrd(int* md, int* nd, int* pd, int n_wid)
{
	int a;
	int Pvalue = 0;
	int col = blockIdx.x * Block_Size + threadIdx.x;
	int row = blockIdx.y * Block_Size + threadIdx.y;
	
	for (a = 0; a < n_wid; a++)
	{
		Pvalue +=(md[row * n_wid + a] * nd[a * n_wid + col]);		
	}
	pd[row * n_wid + col] = Pvalue;
}
int main(void)
{
	int N = 4, i, j, sz;
	int A[4][4]; int B[4][4]; int C[4][4];
	int* d_a, * d_b, * d_c;

	printf("given matrix:\n");
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			A[i][j] = 1;
			B[i][j] = 2;
			C[i][j] = 0;
		}
	}
	printf("Matrix A:\n");
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			printf("%d ", A[i][j]);
		}
		printf("\n");
	}
	printf("Matrix B:\n");
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			printf("%d ", B[i][j]);
		}
		printf("\n");
	}
	sz = sizeof(int) * N * N;
	int Grid_Sz;
	Grid_Sz = N / Block_Size;

	hipMalloc((void**)&d_a,sz);
	hipMalloc((void**)&d_b,sz);
	hipMalloc((void**)&d_c,sz);

	hipMemcpy(d_a,A,sz,hipMemcpyHostToDevice);
	hipMemcpy(d_b,B,sz,hipMemcpyHostToDevice);

	dim3 blockDim(Block_Size, Block_Size,1);
	dim3 gridDim(Grid_Sz, Grid_Sz,1);

	RevWrd << <gridDim, blockDim >> > (d_a, d_b, d_c, N);

	hipMemcpy(C, d_c, sz, hipMemcpyDeviceToHost);
	printf("Matrix C:\n");
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}