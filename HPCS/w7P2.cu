#include"hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void convert(char *s, int n)
{
	int id,r=0,k=0;
	id = threadIdx.x;
	int z;
	z=s[id];
	while(z>0)
		{
			r = z%10;
			z=z/10;
			k = k*10+r;
		}
	s[id]=(char)k;


}

int main(void)
{
	int n,i;
	char s[100];
	
	printf("Enter the string.\n");
	scanf("%s\n",s);

	n = strlen(s);
	//printf("%d",n);

	int size;
	size = sizeof(char);
	char *d_s;

	hipMalloc((void**)&d_s,n*size);	
	hipMemcpy(d_s,s,n*size,hipMemcpyHostToDevice);

	convert<<<1,n>>>(d_s,n);

	hipMemcpy(s,d_s,n*size,hipMemcpyDeviceToHost);
	
	printf("\n");
	for(i=0;i<n;i++)
	printf("%c",s[i]);

	hipFree(d_s);
	return 0;
}