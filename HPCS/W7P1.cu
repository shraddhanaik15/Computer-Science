#include"hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>


__global__ void Toggle(char *a, char *b, int n)
{
	int tid;
	tid = threadIdx.x;
	if(a[tid]>='A' && a[tid]<='Z')
	{
	b[tid]=a[tid]+32;
	}
	else
	{
	b[tid]=a[tid]-32;
	}
}

int main()
{
	char a[100],b[100];

	int i,n,size;
	char *d_a, *d_b;

	printf("\nEnter the string\n");
	scanf("%s",a);

	n = strlen(a);
	printf("\nNo of charcaters is\t%d", n);

	
	size = sizeof(char);
	printf("\nSize is \t%d\n", size);

	hipMalloc((void **)&d_a,n*size);
	hipMalloc((void **)&d_b,n*size);
	

	hipMemcpy(d_a,a,n*size,hipMemcpyHostToDevice);
	

	Toggle<<<1,n>>>(d_a,d_b,n);

	hipMemcpy(b,d_b,n*size,hipMemcpyDeviceToHost);

	
		printf("\nToggled string is \n");
		for(i=0;i<n;i++)
		printf("%c",b[i]);
	
	hipFree(d_a);
	hipFree(d_b);
	
	return 0;


}