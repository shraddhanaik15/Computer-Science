#include"hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>


__global__ void copy(char *a, char *b, int n,int m)
{
	int tid;
	tid = threadIdx.x;
	int i=0;
	while(i<m)
	{
	b[i*n+tid]=a[tid];
	i++;
	}
	
}

int main()
{
	char a[100],b[100];

	int i,n,m,size;
	char *d_a, *d_b;

	printf("\nEnter the string\n");
	scanf("%s", a);

	printf("How many times you want to repeat the string?\n");
	scanf("%d", &m);

	n = strlen(a);
	printf("\nNo of characters is %d\t", n);

	
	size = sizeof(char);
	printf("\nSize is \t%d\n", size);

	hipMalloc((void **)&d_a,n*size);
	hipMalloc((void **)&d_b,n*m*size);
	

	hipMemcpy(d_a,a,n*size,hipMemcpyHostToDevice);
	

	copy<<<1,n>>>(d_a,d_b,n,m);

	hipMemcpy(b,d_b,n*m*size,hipMemcpyDeviceToHost);

		printf("\nRepeated string is \n");
		for(i=0;i<n*m;i++)
		printf("%c",b[i]);
	
	hipFree(d_a);
	hipFree(d_b);
	
	return 0;


}